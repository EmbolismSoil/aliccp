#include "hip/hip_runtime.h"

#define EIGEN_USE_GPU
#include "field_select_kernel.h"
#include <hip/hip_runtime.h>
#include <stdint.h>

__global__ void
target_field_mask(unsigned int const nx,
                  unsigned int const ny,
                  unsigned int const nxy,
                  long long int const* target_field,
                  long long int const* field_ids,
                  int* counts,
                  int* conds,
                  int* mask)
{
    extern __shared__ int local_results[];
    long long int idx = blockDim.x * blockIdx.x + threadIdx.x;
    long long int idy = blockDim.y * blockIdx.y + threadIdx.y;
    long long int tid = nx * idy + idx;

    if (tid >= nxy || idy >= ny || idx >= nx) {
        return;
    }

    // 计算mask
    mask[tid] = field_ids[tid] == target_field[0];
    local_results[nx * threadIdx.y + threadIdx.x] = mask[tid];
    __syncthreads();

    atomicAdd(counts, mask[tid]); //统计个数

    tid = nx * threadIdx.y + threadIdx.x;

    // reduce mask
    for (long long int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (threadIdx.x < stride) {
            local_results[tid] = max(local_results[tid], local_results[tid + stride]);
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        atomicMax(&conds[idy], local_results[tid]);
    }
}

__global__ void
where(unsigned int const nx,
      unsigned int const ny,
      unsigned int const nxy,
      int const* mask,
      long long int const* feat_ids,
      float const* feat_values,
      int const* conds,
      int* counts,
      long long int* output_indices)
{
    unsigned int const idx = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int const idy = blockDim.y * blockIdx.y + threadIdx.y;
    unsigned int const tid = nx * idy + idx;

    if (tid >= nxy || idy >= ny || idx >= nx) {
        return;
    }

    if (mask[tid]) {
        int n = atomicSub(counts, 1);
        output_indices[2 * n] = (long long int)idy;
        output_indices[2 * n + 1] = (long long int)idx;
    }

    if (idx == 0 && conds[idy] == 0) {
        int n = atomicSub(counts, 1);
        output_indices[2 * n] = (long long int)idy;
        output_indices[2 * n + 1] = (long long int)idx;
    }
}

__global__ void
select_feat(unsigned int const nx,
            unsigned int const ny,
            unsigned int const nxy,
            long long int const counts,
            int const* conds,
            long long int const* indices,
            long long int const* feat_ids,
            float const* feat_values,
            long long int* ids,
            float* values)
{
    auto const idx = blockDim.x * blockIdx.x + threadIdx.x;
    auto const idy = blockDim.y * blockIdx.y + threadIdx.y;
    auto const tid = idy + idx;

    if (tid > counts) {
        return;
    }

    auto const y = indices[tid * 2];
    auto const x = indices[tid * 2 + 1];

    if (conds[y] == 0) {
        ids[tid] = 0;
        values[tid] = 0;
    } else {
        ids[tid] = feat_ids[nx * y + x];
        values[tid] = feat_values[nx * y + x];
    }
}
FieldSelectFuctor<Eigen::GpuDevice>::FieldSelectFuctor(Eigen::GpuDevice const& device)
    : device_(device)
{}

template<typename Func, typename... Args>
static void
luanch_cuda_kernel(Func kernel,
                   hipStream_t const& stream,
                   unsigned int const nx,
                   unsigned int const ny,
                   unsigned int const nxy,
                   int smem,
                   Args&&... args)
{
    int blocksize = 0;
    int gridsize = 0;

    hipOccupancyMaxPotentialBlockSize(&gridsize, &blocksize, kernel);
    dim3 block(blocksize, 1);
    dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);

    if (smem < 0) {
        smem = blocksize * sizeof(int);
    }

    kernel<<<grid, block, smem, stream>>>(nx, ny, nxy, std::forward<Args>(args)...);
}

void
FieldSelectFuctor<Eigen::GpuDevice>::launch_target_field_mask(const unsigned int nx,
                                                              const unsigned int ny,
                                                              const unsigned int nxy,
                                                              const long long int* target_field,
                                                              const long long int* field_ids,
                                                              int* counts,
                                                              int* conds,
                                                              int* mask,
                                                              int& cpu_counts)
{
    cpu_counts = 0;
    int* cpu_counts_portable = nullptr;
    int* cpu_conds = nullptr;
    hipHostAlloc(&cpu_conds, sizeof(int) * ny, hipHostMallocDefault);
    hipHostAlloc(&cpu_counts_portable, sizeof(int), hipHostMallocDefault);

    assert(cpu_conds);
    assert(cpu_counts_portable);

    auto const& stream = device_.stream();
    hipMemsetAsync(counts, 0, sizeof(int), stream);
    hipMemsetAsync(conds, 0, ny * sizeof(int), stream);
    luanch_cuda_kernel(
        target_field_mask, stream, nx, ny, nxy, -1, target_field, field_ids, counts, conds, mask);
    hipMemcpyAsync(cpu_counts_portable, counts, sizeof(int), hipMemcpyDeviceToHost, stream);
    hipMemcpyAsync(cpu_conds, conds, sizeof(int) * ny, hipMemcpyDeviceToHost, stream);
    hipStreamSynchronize(stream); // wait for done

    cpu_counts =
        std::accumulate(cpu_conds, cpu_conds + ny, *cpu_counts_portable, [](int const lhs, int const rhs) {
            return lhs + !rhs;
        });

    *cpu_counts_portable = cpu_counts - 1;
    hipMemcpyAsync(counts, cpu_counts_portable, sizeof(int), hipMemcpyHostToDevice, stream);
    hipStreamSynchronize(stream); // wait for done
    hipHostFree(cpu_conds);
    hipHostFree(cpu_counts_portable);
}

void
FieldSelectFuctor<Eigen::GpuDevice>::luanch_select_feat(const unsigned int nx,
                                                        const unsigned int ny,
                                                        const unsigned int nxy,
                                                        const int* mask,
                                                        const long long int* feat_ids,
                                                        const float* feat_values,
                                                        const int* conds,
                                                        int* counts,
                                                        long long int* output_indices,
                                                        long long int* output_feat_ids,
                                                        float* output_feat_values)
{
    auto const& stream = device_.stream();
    int* cpu_counts = nullptr;
    hipHostAlloc(&cpu_counts, sizeof(int), hipHostMallocDefault);
    assert(cpu_counts);
    hipMemcpyAsync(cpu_counts, counts, sizeof(int), hipMemcpyDeviceToHost, stream);

    luanch_cuda_kernel(
        where, stream, nx, ny, nxy, 0, mask, feat_ids, feat_values, conds, counts, output_indices);
    hipStreamSynchronize(stream);

    *cpu_counts = *cpu_counts + 1;
    long long int* cpu_indices = nullptr;
    hipHostAlloc(&cpu_indices, sizeof(long long int) * *cpu_counts * 2, hipHostMallocDefault);
    assert(cpu_indices);
    hipMemcpy(cpu_indices, output_indices, sizeof(long long int) * *cpu_counts * 2, hipMemcpyDeviceToHost);

    std::vector<uint32_t> indices(*cpu_counts, 0);
    std::iota(indices.begin(), indices.end(), 0);
    std::sort(indices.begin(), indices.end(), [cpu_indices](uint32_t const lhs, uint32_t const rhs) {
        if (cpu_indices[2 * lhs] < cpu_indices[2 * rhs])
            return true;
        else if (cpu_indices[2 * lhs] == cpu_indices[2 * rhs])
            return cpu_indices[2 * lhs + 1] < cpu_indices[2 * rhs + 1];
        else
            return false;
    });

    std::vector<long long int> sorted_indices;
    for (auto const i : indices) {
        sorted_indices.push_back(cpu_indices[i * 2]);
        sorted_indices.push_back(cpu_indices[i * 2 + 1]);
    }

    hipMemcpy(output_indices,
               sorted_indices.data(),
               sizeof(long long int) * *cpu_counts * 2,
               hipMemcpyHostToDevice);

    luanch_cuda_kernel(select_feat,
                       stream,
                       nx,
                       (unsigned int)*cpu_counts,
                       (unsigned int)*cpu_counts,
                       0,
                       *cpu_counts,
                       conds,
                       output_indices,
                       feat_ids,
                       feat_values,
                       output_feat_ids,
                       output_feat_values);
    hipStreamSynchronize(stream); // wait for done

    hipFree(cpu_indices);
    hipFree(cpu_counts);
}
